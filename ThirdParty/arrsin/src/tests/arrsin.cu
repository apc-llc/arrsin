#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdio>
#include <cstdlib>
#include <chrono>
#include <cmath>
#include <execution>
#include <random>
#include <vector>
#include "check.h"

#define CUDA_ERR_CHECK(x) \
        do { hipError_t err = x; if (err != hipSuccess) { \
                fprintf (stderr, "Error \"%s\" at %s:%d \n", \
                 hipGetErrorString(err), \
                __FILE__, __LINE__); exit(-1); \
        }} while (0);

using namespace std;

using fptype = FPTYPE;

__global__ void kernel(fptype* x)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	x[i] = sin(x[i]);
}

int main(int argc, char* argv[])
{
	std::mt19937 gen;
	std::uniform_real_distribution<> dist(-1, 1);

	vector<fptype> x(1e8);
	generate(x.begin(), x.end(), [&dist, &gen]()
	{
		return dist(gen);
	});

	int szblock = 128;
	fptype* x_dev = nullptr;
	size_t nblocks = x.size() / szblock;
	nblocks += x.size() % szblock ? 1 : 0;
	CUDA_ERR_CHECK(hipMalloc(&x_dev, szblock * nblocks * sizeof(fptype)));
	CUDA_ERR_CHECK(hipMemcpy(x_dev, x.data(), x.size() * sizeof(fptype), hipMemcpyHostToDevice));

	auto t1 = std::chrono::high_resolution_clock::now();

	kernel<<<nblocks, szblock>>>(x_dev);
	CUDA_ERR_CHECK(hipGetLastError());
	CUDA_ERR_CHECK(hipDeviceSynchronize());

	auto t2 = std::chrono::high_resolution_clock::now();
	
	auto time = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count();
	printf("%f\n", time / 1e6);
	
	CUDA_ERR_CHECK(hipMemcpy(x.data(), x_dev, x.size() * sizeof(fptype), hipMemcpyDeviceToHost));
	CUDA_ERR_CHECK(hipFree(x_dev));

	check(x);

	return 0;
}

// BENCH DEV NVIDIA GeForce 1060M
// BENCH NAME GPU NVIDIA GTX1060M (nvc++ -stdpar)
// BENCH EXEC ./arrsin_nvpar_float
// BENCH SCORE 0.005465
// BENCH CHECK check sum = -3187.200205

// BENCH DEV NVIDIA GeForce 1060M
// BENCH NAME GPU NVIDIA GTX1060M (nvc++ -stdpar)
// BENCH EXEC ./arrsin_nvpar_double
// BENCH SCORE 0.030684
// BENCH CHECK check sum = -3187.199840

// BENCH DEV NVIDIA Tesla V100-SXM2-16GB
// BENCH NAME GPU NVIDIA V100 (nvc++ -stdpar)
// BENCH EXEC ./arrsin_nvpar_float
// BENCH SCORE 0.001142
// BENCH CHECK check sum = -3187.200205

// BENCH DEV NVIDIA Tesla V100-SXM2-16GB
// BENCH NAME GPU NVIDIA V100 (nvc++ -stdpar)
// BENCH EXEC ./arrsin_nvpar_double
// BENCH SCORE 0.002970
// BENCH CHECK check sum = -3187.199840

// BENCH DEV NVIDIA Tesla A100-SXM4-40GB
// BENCH NAME GPU NVIDIA A100 (nvc++ -stdpar)
// BENCH EXEC ./arrsin_nvpar_float
// BENCH SCORE 0.001030
// BENCH CHECK check sum = -3187.200205

// BENCH DEV NVIDIA Tesla A100-SXM4-40GB
// BENCH NAME GPU NVIDIA A100 (nvc++ -stdpar)
// BENCH EXEC ./arrsin_nvpar_double
// BENCH SCORE 0.005156
// BENCH CHECK check sum = -3187.199840

